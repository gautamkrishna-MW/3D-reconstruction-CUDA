//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
//
// generateScene3DGpuImpl_terminate.cu
//
// Code generation for function 'generateScene3DGpuImpl_terminate'
//

// Include files
#include "generateScene3DGpuImpl_terminate.h"
#include "_coder_generateScene3DGpuImpl_mex.h"
#include "generateScene3DGpuImpl_data.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWMemoryManager.hpp"

// Function Definitions
void generateScene3DGpuImpl_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  mwCudaFree(&(*fv_gpu_clone)[0]);
}

void generateScene3DGpuImpl_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  mwMemoryManagerTerminate();
}

// End of code generation (generateScene3DGpuImpl_terminate.cu)
