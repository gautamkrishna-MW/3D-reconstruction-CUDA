#include "hip/hip_runtime.h"
//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
//
// generateScene3DGpuImpl.cu
//
// Code generation for function 'generateScene3DGpuImpl'
//

// Include files
#include "generateScene3DGpuImpl.h"
#include "generateScene3DGpuImpl_data.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWPtxUtils.hpp"
#include "disparitySGMWrapperCuda.hpp"
#include "gpudisparitySGMConfig.hpp"
#include "gpudisparitySGMCostCuda.hpp"
#include "gpudisparitySGMCuda.hpp"
#include "hip/hip_math_constants.h"

// Variable Definitions
static boolean_T c_gpuConstsCopied_generateScene;

// Function Declarations
static __global__ void
generateScene3DGpuImpl_kernel1(const uint8_T rightImage[2073600],
                               const uint8_T leftImage[2073600],
                               uint8_T I2U8[2073600], uint8_T I1U8[2073600]);

static __global__ void generateScene3DGpuImpl_kernel10(
    const real32_T disparityMap[2073600],
    const real32_T pts3DHomogeneousReproject[8294400],
    real32_T pts3DOut[6220800]);

static __global__ void
generateScene3DGpuImpl_kernel11(boolean_T *globalConvergenceFlag,
                                real32_T disparityMap[2073600]);

static __global__ void generateScene3DGpuImpl_kernel2(
    const uint8_T I2U8[2073600], const uint8_T I1U8[2073600],
    uint8_T rightRect[2073600], uint8_T leftRect[2073600], real32_T a_dim0,
    real32_T a_dim1, real32_T a_dim2, real32_T a_dim3, real32_T a_dim4,
    real32_T a_dim5, real32_T a_dim6, real32_T a_dim7, real32_T a_dim8,
    real32_T b_a_dim0, real32_T b_a_dim1, real32_T b_a_dim2, real32_T b_a_dim3,
    real32_T b_a_dim4, real32_T b_a_dim5, real32_T b_a_dim6, real32_T b_a_dim7,
    real32_T b_a_dim8);

static __global__ void
generateScene3DGpuImpl_kernel3(const real32_T rightFiltered[2073600],
                               const real32_T leftFiltered[2073600],
                               uint8_T I2U8[2073600], uint8_T I1U8[2073600]);

static __global__ void
generateScene3DGpuImpl_kernel4(const uint8_T I2U8[2073600],
                               const uint8_T I1U8[2073600], uint8_T I4[2073600],
                               uint8_T I3[2073600]);

static __global__ void
generateScene3DGpuImpl_kernel5(const real32_T leftFiltered[2073600],
                               real32_T disparityMap[2073600]);

static __global__ void
generateScene3DGpuImpl_kernel6(real32_T disparityMap[2073600]);

static __global__ void generateScene3DGpuImpl_kernel7(int16_T yCoord[2073600],
                                                      int16_T xCoord[2073600]);

static __global__ void generateScene3DGpuImpl_kernel8(
    const real32_T disparityMap[2073600], const int16_T yCoord[2073600],
    const int16_T xCoord[2073600], real32_T pts3DHomogeneous[8294400]);

static __global__ void
generateScene3DGpuImpl_kernel9(const real32_T fv[16],
                               const real32_T pts3DHomogeneous[8294400],
                               real32_T pts3DHomogeneousReproject[8294400]);

static __global__ void stencilKernel(const uint8_T input[2073600],
                                     uint8_T paddingValue,
                                     real32_T output[2073600]);

// Function Definitions
static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel1(
    const uint8_T rightImage[2073600], const uint8_T leftImage[2073600],
    uint8_T I2U8[2073600], uint8_T I1U8[2073600])
{
  uint64_T threadId;
  int32_T colIter;
  int32_T rowIter;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int32_T>(threadId % 1080ULL);
  colIter = static_cast<int32_T>((threadId - static_cast<uint64_T>(rowIter)) /
                                 1080ULL);
  if ((colIter < 1920) && (rowIter < 1080)) {
    real32_T b_y;
    real32_T c_y;
    real32_T colIterR;
    real32_T dU;
    real32_T inpColR;
    real32_T inpRowR;
    real32_T outColL;
    real32_T r_2;
    real32_T radMultL;
    real32_T radMultR;
    real32_T rowIterL;
    real32_T rowIterR;
    real32_T sumIntensities;
    real32_T val;
    real32_T xy2;
    real32_T y;
    uint8_T i4L;
    //         %% Left Image
    sumIntensities =
        ((static_cast<real32_T>(colIter) + 1.0F) - 919.169312F) / 1266.84326F;
    rowIterL =
        ((static_cast<real32_T>(rowIter) + 1.0F) - 546.094849F) / 1267.10303F;
    //  r = sqrt(rowIter*rowIter + colIter*colIter);
    r_2 = rowIterL * rowIterL + sumIntensities * sumIntensities;
    val = r_2 * r_2;
    //  Radial
    radMultL = ((-0.136658221F * r_2 + 1.0F) + 0.168160394F * val) +
               -0.0391957462F * (val * r_2);
    //  Tangential
    xy2 = 2.0F * rowIterL * sumIntensities;
    dU = xy2 * -0.00227867416F;
    c_y = -0.00232284726F * (r_2 + 2.0F * rowIterL * rowIterL);
    inpColR = (rowIterL * radMultL + xy2 * -0.00227867416F) +
              -0.00232284726F * (r_2 + 2.0F * rowIterL * rowIterL);
    b_y = xy2 * -0.00232284726F;
    y = -0.00227867416F * (r_2 + 2.0F * sumIntensities * sumIntensities);
    inpRowR = (sumIntensities * radMultL + xy2 * -0.00232284726F) +
              -0.00227867416F * (r_2 + 2.0F * sumIntensities * sumIntensities);
    //         %% Right Image
    colIterR =
        ((static_cast<real32_T>(colIter) + 1.0F) - 1040.93506F) / 1272.84253F;
    rowIterR =
        ((static_cast<real32_T>(rowIter) + 1.0F) - 561.620178F) / 1272.45984F;
    //  r = sqrt(rowIter*rowIter + colIter*colIter);
    r_2 = rowIterR * rowIterR + colIterR * colIterR;
    val = r_2 * r_2;
    //  Radial
    radMultR = ((-0.131053701F * r_2 + 1.0F) + 0.161947727F * val) +
               -0.0327948183F * (val * r_2);
    //  Tangential
    xy2 = 2.0F * rowIterR * colIterR;
    //         %% Map back
    val = ((sumIntensities * radMultL + b_y) + y) * 1266.84326F;
    outColL = inpRowR * 1266.84326F + 919.169312F;
    c_y = ((rowIterL * radMultL + dU) + c_y) * 1267.10303F;
    inpColR = inpColR * 1267.10303F + 546.094849F;
    b_y = ((colIterR * radMultR + xy2 * 0.00366475782F) +
           0.000479186914F * (r_2 + 2.0F * colIterR * colIterR)) *
          1272.84253F;
    inpRowR = ((colIterR * radMultR + xy2 * 0.00366475782F) +
               0.000479186914F * (r_2 + 2.0F * colIterR * colIterR)) *
                  1272.84253F +
              1040.93506F;
    y = ((rowIterR * radMultR + xy2 * 0.000479186914F) +
         0.00366475782F * (r_2 + 2.0F * rowIterR * rowIterR)) *
        1272.45984F;
    sumIntensities = ((rowIterR * radMultR + xy2 * 0.000479186914F) +
                      0.00366475782F * (r_2 + 2.0F * rowIterR * rowIterR)) *
                         1272.45984F +
                     561.620178F;
    //  Interpolation
    if (c_y + 546.094849F >= 1080.0F) {
      inpColR = 1079.0F;
    } else if (c_y + 546.094849F < 1.0F) {
      inpColR = 1.0F;
    }
    if (y + 561.620178F >= 1080.0F) {
      sumIntensities = 1079.0F;
    } else if (y + 561.620178F < 1.0F) {
      sumIntensities = 1.0F;
    }
    if (val + 919.169312F >= 1920.0F) {
      outColL = 1919.0F;
    } else if (val + 919.169312F < 1.0F) {
      outColL = 1.0F;
    }
    if (b_y + 1040.93506F >= 1920.0F) {
      inpRowR = 1919.0F;
    } else if (b_y + 1040.93506F < 1.0F) {
      inpRowR = 1.0F;
    }
    //  Interpolation for left image
    dU = inpColR - floorf(inpColR);
    val = outColL - floorf(outColL);
    val = roundf(
        ((static_cast<real32_T>(
              leftImage[static_cast<int32_T>(floorf(inpColR) +
                                             1080.0 * (floorf(outColL) - 1.0)) -
                        1]) *
              (1.0F - dU) * (1.0F - val) +
          static_cast<real32_T>(leftImage[static_cast<int32_T>(
              floorf(inpColR) + 1080.0 * (floorf(outColL) - 1.0))]) *
              dU * (1.0F - val)) +
         static_cast<real32_T>(
             leftImage[static_cast<int32_T>(floorf(inpColR) +
                                            1080.0 * floorf(outColL)) -
                       1]) *
             (1.0F - dU) * val) +
        static_cast<real32_T>(leftImage[static_cast<int32_T>(
            floorf(inpColR) + 1080.0 * floorf(outColL))]) *
            dU * val);
    if (val < 256.0F) {
      if (val >= 0.0F) {
        i4L = static_cast<uint8_T>(val);
      } else {
        i4L = 0U;
      }
    } else {
      i4L = MAX_uint8_T;
    }
    I1U8[rowIter + 1080 * colIter] = i4L;
    //  Interpolation for right image
    dU = sumIntensities - floorf(sumIntensities);
    val = inpRowR - floorf(inpRowR);
    val = roundf(
        ((static_cast<real32_T>(
              rightImage[static_cast<int32_T>(floorf(sumIntensities) +
                                              1080.0 *
                                                  (floorf(inpRowR) - 1.0)) -
                         1]) *
              (1.0F - dU) * (1.0F - val) +
          static_cast<real32_T>(rightImage[static_cast<int32_T>(
              floorf(sumIntensities) + 1080.0 * (floorf(inpRowR) - 1.0))]) *
              dU * (1.0F - val)) +
         static_cast<real32_T>(
             rightImage[static_cast<int32_T>(floorf(sumIntensities) +
                                             1080.0 * floorf(inpRowR)) -
                        1]) *
             (1.0F - dU) * val) +
        static_cast<real32_T>(rightImage[static_cast<int32_T>(
            floorf(sumIntensities) + 1080.0 * floorf(inpRowR))]) *
            dU * val);
    if (val < 256.0F) {
      if (val >= 0.0F) {
        i4L = static_cast<uint8_T>(val);
      } else {
        i4L = 0U;
      }
    } else {
      i4L = MAX_uint8_T;
    }
    I2U8[rowIter + 1080 * colIter] = i4L;
  }
}

static __global__
    __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel10(
        const real32_T disparityMap[2073600],
        const real32_T pts3DHomogeneousReproject[8294400],
        real32_T pts3DOut[6220800])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 2073600) {
    real32_T val;
    val = pts3DHomogeneousReproject[b_index + 6220800];
    pts3DOut[b_index] = pts3DHomogeneousReproject[b_index] / val;
    pts3DOut[b_index + 2073600] =
        pts3DHomogeneousReproject[b_index + 2073600] / val;
    pts3DOut[b_index + 4147200] =
        pts3DHomogeneousReproject[b_index + 4147200] / val;
    if (disparityMap[b_index] == -3.402823466E+38F) {
      pts3DOut[b_index] = HIP_NAN_F;
      pts3DOut[b_index + 2073600] = HIP_NAN_F;
      pts3DOut[b_index + 4147200] = HIP_NAN_F;
    }
  }
}

static __global__
    __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel11(
        boolean_T *globalConvergenceFlag, real32_T disparityMap[2073600])
{
  uint64_T threadId;
  int32_T colIter;
  int32_T rowIter;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int32_T>(threadId % 1078ULL);
  colIter = static_cast<int32_T>((threadId - static_cast<uint64_T>(rowIter)) /
                                 1078ULL);
  if ((colIter < 1919) && (rowIter < 1078) &&
      isinf(disparityMap[(rowIter + 1080 * colIter) + 1])) {
    real_T countIntensities;
    real32_T sumIntensities;
    //  Thread checks for NaN in N-8 and if found one, runs the
    //  algorithm one more time.
    *globalConvergenceFlag = true;
    sumIntensities = 0.0F;
    countIntensities = 0.0;
    for (int32_T b_index{0}; b_index < 3; b_index++) {
      if (!isinf(disparityMap[rowIter + 1080 * ((colIter + b_index) - 1)])) {
        sumIntensities +=
            disparityMap[rowIter + 1080 * ((colIter + b_index) - 1)];
        countIntensities++;
      }
      if (!isinf(
              disparityMap[(rowIter + 1080 * ((colIter + b_index) - 1)) + 1])) {
        sumIntensities +=
            disparityMap[(rowIter + 1080 * ((colIter + b_index) - 1)) + 1];
        countIntensities++;
      }
      if (!isinf(
              disparityMap[(rowIter + 1080 * ((colIter + b_index) - 1)) + 2])) {
        sumIntensities +=
            disparityMap[(rowIter + 1080 * ((colIter + b_index) - 1)) + 2];
        countIntensities++;
      }
    }
    if (countIntensities > 0.0) {
      disparityMap[(rowIter + 1080 * colIter) + 1] =
          sumIntensities / static_cast<real32_T>(countIntensities);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel2(
    const uint8_T I2U8[2073600], const uint8_T I1U8[2073600],
    uint8_T rightRect[2073600], uint8_T leftRect[2073600], real32_T a_dim0,
    real32_T a_dim1, real32_T a_dim2, real32_T a_dim3, real32_T a_dim4,
    real32_T a_dim5, real32_T a_dim6, real32_T a_dim7, real32_T a_dim8,
    real32_T b_a_dim0, real32_T b_a_dim1, real32_T b_a_dim2, real32_T b_a_dim3,
    real32_T b_a_dim4, real32_T b_a_dim5, real32_T b_a_dim6, real32_T b_a_dim7,
    real32_T b_a_dim8)
{
  __shared__ real32_T a_shared[9];
  __shared__ real32_T b_a_shared[9];
  uint64_T threadId;
  int32_T colIter;
  int32_T rowIter;
  if (mwGetThreadIndexWithinBlock() == 0) {
    b_a_shared[0] = b_a_dim0;
    b_a_shared[1] = b_a_dim1;
    b_a_shared[2] = b_a_dim2;
    b_a_shared[3] = b_a_dim3;
    b_a_shared[4] = b_a_dim4;
    b_a_shared[5] = b_a_dim5;
    b_a_shared[6] = b_a_dim6;
    b_a_shared[7] = b_a_dim7;
    b_a_shared[8] = b_a_dim8;
  }
  __syncthreads();
  if (mwGetThreadIndexWithinBlock() == 0) {
    a_shared[0] = a_dim0;
    a_shared[1] = a_dim1;
    a_shared[2] = a_dim2;
    a_shared[3] = a_dim3;
    a_shared[4] = a_dim4;
    a_shared[5] = a_dim5;
    a_shared[6] = a_dim6;
    a_shared[7] = a_dim7;
    a_shared[8] = a_dim8;
  }
  __syncthreads();
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  colIter = static_cast<int32_T>(threadId % 1920ULL);
  rowIter = static_cast<int32_T>((threadId - static_cast<uint64_T>(colIter)) /
                                 1920ULL);
  if ((rowIter < 1080) && (colIter < 1920)) {
    real32_T inpPointL[3];
    real32_T dU;
    real32_T inpColR;
    real32_T inpRowR;
    real32_T sumIntensities;
    real32_T val;
    uint8_T i1L;
    uint8_T i1R;
    uint8_T i2L;
    uint8_T i2R;
    uint8_T i3L;
    uint8_T i3R;
    uint8_T i4L;
    uint8_T i4R;
    //  Left Image
    for (int32_T i{0}; i < 3; i++) {
      inpPointL[i] =
          (a_shared[i] * (static_cast<real32_T>(colIter) + 1.0F) +
           a_shared[i + 3] * (static_cast<real32_T>(rowIter) + 1.0F)) +
          a_shared[i + 6];
    }
    sumIntensities = inpPointL[0] / inpPointL[2];
    val = inpPointL[1] / inpPointL[2];
    //  Right Image
    for (int32_T i{0}; i < 3; i++) {
      inpPointL[i] =
          (b_a_shared[i] * (static_cast<real32_T>(colIter) + 1.0F) +
           b_a_shared[i + 3] * (static_cast<real32_T>(rowIter) + 1.0F)) +
          b_a_shared[i + 6];
    }
    inpColR = inpPointL[0] / inpPointL[2];
    inpRowR = inpPointL[1] / inpPointL[2];
    //  Bounds Check Left
    if ((val > 1.0F) && (val <= 1080.0F) && (sumIntensities > 1.0F) &&
        (sumIntensities <= 1920.0F)) {
      i1L = I1U8[static_cast<int32_T>(floorf(val) +
                                      1080.0 * (floorf(sumIntensities) - 1.0)) -
                 1];
    } else {
      i1L = 0U;
    }
    if ((val > 1.0F) && (val <= 1079.0F) && (sumIntensities > 1.0F) &&
        (sumIntensities <= 1920.0F)) {
      i2L = I1U8[static_cast<int32_T>(floorf(val) +
                                      1080.0 * (floorf(sumIntensities) - 1.0))];
    } else {
      i2L = 0U;
    }
    if ((val > 1.0F) && (val <= 1080.0F) && (sumIntensities > 1.0F) &&
        (sumIntensities <= 1919.0F)) {
      i3L = I1U8[static_cast<int32_T>(floorf(val) +
                                      1080.0 * floorf(sumIntensities)) -
                 1];
    } else {
      i3L = 0U;
    }
    if ((val > 1.0F) && (val <= 1079.0F) && (sumIntensities > 1.0F) &&
        (sumIntensities <= 1919.0F)) {
      i4L = I1U8[static_cast<int32_T>(floorf(val) +
                                      1080.0 * floorf(sumIntensities))];
    } else {
      i4L = 0U;
    }
    //  Bounds Check Right
    if ((inpRowR > 1.0F) && (inpRowR <= 1080.0F) && (inpColR > 1.0F) &&
        (inpColR <= 1920.0F)) {
      i1R = I2U8[static_cast<int32_T>(floorf(inpRowR) +
                                      1080.0 * (floorf(inpColR) - 1.0)) -
                 1];
    } else {
      i1R = 0U;
    }
    if ((inpRowR > 1.0F) && (inpRowR <= 1079.0F) && (inpColR > 1.0F) &&
        (inpColR <= 1920.0F)) {
      i2R = I2U8[static_cast<int32_T>(floorf(inpRowR) +
                                      1080.0 * (floorf(inpColR) - 1.0))];
    } else {
      i2R = 0U;
    }
    if ((inpRowR > 1.0F) && (inpRowR <= 1080.0F) && (inpColR > 1.0F) &&
        (inpColR <= 1919.0F)) {
      i3R = I2U8[static_cast<int32_T>(floorf(inpRowR) +
                                      1080.0 * floorf(inpColR)) -
                 1];
    } else {
      i3R = 0U;
    }
    if ((inpRowR > 1.0F) && (inpRowR <= 1079.0F) && (inpColR > 1.0F) &&
        (inpColR <= 1919.0F)) {
      i4R = I2U8[static_cast<int32_T>(floorf(inpRowR) +
                                      1080.0 * floorf(inpColR))];
    } else {
      i4R = 0U;
    }
    //  Interpolation for left image
    dU = val - floorf(val);
    val = sumIntensities - floorf(sumIntensities);
    val = roundf(((static_cast<real32_T>(i1L) * (1.0F - dU) * (1.0F - val) +
                   static_cast<real32_T>(i2L) * dU * (1.0F - val)) +
                  static_cast<real32_T>(i3L) * (1.0F - dU) * val) +
                 static_cast<real32_T>(i4L) * dU * val);
    if (val < 256.0F) {
      if (val >= 0.0F) {
        i4L = static_cast<uint8_T>(val);
      } else {
        i4L = 0U;
      }
    } else if (val >= 256.0F) {
      i4L = MAX_uint8_T;
    } else {
      i4L = 0U;
    }
    leftRect[rowIter + 1080 * colIter] = i4L;
    //  Interpolation for right image
    dU = inpRowR - floorf(inpRowR);
    val = inpColR - floorf(inpColR);
    val = roundf(((static_cast<real32_T>(i1R) * (1.0F - dU) * (1.0F - val) +
                   static_cast<real32_T>(i2R) * dU * (1.0F - val)) +
                  static_cast<real32_T>(i3R) * (1.0F - dU) * val) +
                 static_cast<real32_T>(i4R) * dU * val);
    if (val < 256.0F) {
      if (val >= 0.0F) {
        i4L = static_cast<uint8_T>(val);
      } else {
        i4L = 0U;
      }
    } else {
      i4L = MAX_uint8_T;
    }
    rightRect[rowIter + 1080 * colIter] = i4L;
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel3(
    const real32_T rightFiltered[2073600], const real32_T leftFiltered[2073600],
    uint8_T I2U8[2073600], uint8_T I1U8[2073600])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 2073600) {
    real32_T val;
    //  Compute disparity
    val = leftFiltered[b_index] * 255.0F;
    if (val < 0.0F) {
      I1U8[b_index] = 0U;
    } else if (val > 255.0F) {
      I1U8[b_index] = MAX_uint8_T;
    } else {
      I1U8[b_index] = static_cast<uint8_T>(val + 0.5F);
    }
    val = rightFiltered[b_index] * 255.0F;
    if (val < 0.0F) {
      I2U8[b_index] = 0U;
    } else if (val > 255.0F) {
      I2U8[b_index] = MAX_uint8_T;
    } else {
      I2U8[b_index] = static_cast<uint8_T>(val + 0.5F);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel4(
    const uint8_T I2U8[2073600], const uint8_T I1U8[2073600],
    uint8_T I4[2073600], uint8_T I3[2073600])
{
  uint64_T threadId;
  int32_T b_index;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId % 1920ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(b_index)) /
                           1920ULL);
  if ((i < 1080) && (b_index < 1920)) {
    I3[b_index + 1920 * i] = I1U8[i + 1080 * b_index];
    I4[b_index + 1920 * i] = I2U8[i + 1080 * b_index];
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel5(
    const real32_T leftFiltered[2073600], real32_T disparityMap[2073600])
{
  uint64_T threadId;
  int32_T b_index;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId % 1080ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(b_index)) /
                           1080ULL);
  if ((i < 1920) && (b_index < 1080)) {
    disparityMap[b_index + 1080 * i] = leftFiltered[i + 1920 * b_index];
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel6(
    real32_T disparityMap[2073600])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 2073600) {
    real32_T val;
    val = disparityMap[b_index];
    if (val == 0.0F) {
      disparityMap[b_index] = -HIP_INF_F;
    } else if (val == 39.0F) {
      disparityMap[b_index] = HIP_INF_F;
    }
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel7(
    int16_T yCoord[2073600], int16_T xCoord[2073600])
{
  uint64_T threadId;
  int32_T b_index;
  int32_T colIter;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  colIter = static_cast<int32_T>(threadId % 1080ULL);
  b_index = static_cast<int32_T>((threadId - static_cast<uint64_T>(colIter)) /
                                 1080ULL);
  if ((b_index < 1920) && (colIter < 1080)) {
    xCoord[colIter + 1080 * b_index] = static_cast<int16_T>(b_index + 1);
    yCoord[colIter + 1080 * b_index] = static_cast<int16_T>(colIter + 1);
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel8(
    const real32_T disparityMap[2073600], const int16_T yCoord[2073600],
    const int16_T xCoord[2073600], real32_T pts3DHomogeneous[8294400])
{
  uint64_T threadId;
  int32_T b_index;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId);
  if (b_index < 2073600) {
    //  Stack points
    pts3DHomogeneous[b_index] = xCoord[b_index];
    pts3DHomogeneous[b_index + 2073600] = yCoord[b_index];
    pts3DHomogeneous[b_index + 4147200] = disparityMap[b_index];
    pts3DHomogeneous[b_index + 6220800] = 1.0F;
  }
}

static __global__ __launch_bounds__(512, 1) void generateScene3DGpuImpl_kernel9(
    const real32_T fv[16], const real32_T pts3DHomogeneous[8294400],
    real32_T pts3DHomogeneousReproject[8294400])
{
  uint64_T threadId;
  int32_T b_index;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_index = static_cast<int32_T>(threadId % 4ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(b_index)) / 4ULL);
  if ((i < 2073600) && (b_index < 4)) {
    pts3DHomogeneousReproject[i + 2073600 * b_index] =
        ((pts3DHomogeneous[i] * fv[b_index << 2] +
          pts3DHomogeneous[i + 2073600] * fv[(b_index << 2) + 1]) +
         pts3DHomogeneous[i + 4147200] * fv[(b_index << 2) + 2]) +
        pts3DHomogeneous[i + 6220800] * fv[(b_index << 2) + 3];
  }
}

static __global__
    __launch_bounds__(256, 1) void stencilKernel(const uint8_T input[2073600],
                                                 uint8_T paddingValue,
                                                 real32_T output[2073600])
{
  int32_T b_workItemGlobalOutputElemDimId;
  int32_T sum;
  int32_T workGroupIdTmp;
  int32_T workItemGlobalOutputElemDimIdx;
  sum = mwGetThreadIndexWithinBlock();
  workGroupIdTmp = mwGetBlockIndex();
  workItemGlobalOutputElemDimIdx = sum % 16 + ((workGroupIdTmp % 68) << 4);
  workGroupIdTmp /= 68;
  b_workItemGlobalOutputElemDimId = sum / 16 + (workGroupIdTmp << 4);
  if ((workItemGlobalOutputElemDimIdx < 1080) &&
      (b_workItemGlobalOutputElemDimId < 1920)) {
    uint8_T window[121];
    for (int32_T windowIdx{0}; windowIdx < 11; windowIdx++) {
      workGroupIdTmp = (b_workItemGlobalOutputElemDimId + windowIdx) - 5;
      for (int32_T b_windowIdx{0}; b_windowIdx < 11; b_windowIdx++) {
        sum = (workItemGlobalOutputElemDimIdx + b_windowIdx) - 5;
        if ((sum >= 0) && (sum < 1080) && (workGroupIdTmp >= 0) &&
            (workGroupIdTmp < 1920)) {
          window[b_windowIdx + 11 * windowIdx] =
              input[sum + 1080 * workGroupIdTmp];
        } else {
          window[b_windowIdx + 11 * windowIdx] = paddingValue;
        }
      }
    }
    sum = 0;
    for (workGroupIdTmp = 0; workGroupIdTmp < 121; workGroupIdTmp++) {
      sum += window[workGroupIdTmp];
    }
    output[workItemGlobalOutputElemDimIdx +
           1080 * b_workItemGlobalOutputElemDimId] =
        static_cast<real32_T>(sum) / 30855.0F;
  }
}

void generateScene3DGpuImpl(const uint8_T leftImage[2073600],
                            const uint8_T rightImage[2073600],
                            uint8_T leftRect[2073600],
                            uint8_T rightRect[2073600],
                            real32_T disparityMap[2073600],
                            real32_T pts3DOut[6220800])
{
  static const real32_T fv[16]{
      1.0F, 0.0F, 0.0F, -919.169312F, 0.0F, 1.0F, 0.0F,         -546.094849F,
      0.0F, 0.0F, 0.0F, 1266.84326F,  0.0F, 0.0F, 0.220501065F, 26.8494778F};
  real32_T(*gpu_pts3DHomogeneous)[8294400];
  real32_T(*gpu_pts3DHomogeneousReproject)[8294400];
  real32_T(*gpu_pts3DOut)[6220800];
  real32_T(*gpu_disparityMap)[2073600];
  real32_T(*gpu_leftFiltered)[2073600];
  real32_T(*gpu_rightFiltered)[2073600];
  int16_T(*gpu_xCoord)[2073600];
  int16_T(*gpu_yCoord)[2073600];
  uint8_T(*gpu_gpuWorkspace)[503884800];
  uint8_T(*gpu_I1U8)[2073600];
  uint8_T(*gpu_I2U8)[2073600];
  uint8_T(*gpu_I3)[2073600];
  uint8_T(*gpu_I4)[2073600];
  uint8_T(*gpu_leftImage)[2073600];
  uint8_T(*gpu_leftRect)[2073600];
  uint8_T(*gpu_rightImage)[2073600];
  uint8_T(*gpu_rightRect)[2073600];
  boolean_T globalConvergenceFlag;
  boolean_T *gpu_globalConvergenceFlag;
  if (!c_gpuConstsCopied_generateScene) {
    c_gpuConstsCopied_generateScene = true;
    hipMemcpy(*fv_gpu_clone, fv, sizeof(real32_T[16]), hipMemcpyHostToDevice);
  }
  mwCudaMalloc(&gpu_pts3DOut, 24883200ULL);
  mwCudaMalloc(&gpu_pts3DHomogeneousReproject, 33177600ULL);
  mwCudaMalloc(&gpu_pts3DHomogeneous, 33177600ULL);
  mwCudaMalloc(&gpu_xCoord, 4147200ULL);
  mwCudaMalloc(&gpu_yCoord, 4147200ULL);
  mwCudaMalloc(&gpu_globalConvergenceFlag, 1ULL);
  mwCudaMalloc(&gpu_disparityMap, 8294400ULL);
  mwCudaMalloc(&gpu_gpuWorkspace, 503884800ULL);
  mwCudaMalloc(&gpu_I3, 2073600ULL);
  mwCudaMalloc(&gpu_I4, 2073600ULL);
  mwCudaMalloc(&gpu_rightFiltered, 8294400ULL);
  mwCudaMalloc(&gpu_leftFiltered, 8294400ULL);
  mwCudaMalloc(&gpu_leftRect, 2073600ULL);
  mwCudaMalloc(&gpu_rightRect, 2073600ULL);
  mwCudaMalloc(&gpu_I1U8, 2073600ULL);
  mwCudaMalloc(&gpu_I2U8, 2073600ULL);
  mwCudaMalloc(&gpu_leftImage, 2073600ULL);
  mwCudaMalloc(&gpu_rightImage, 2073600ULL);
  //  Undistort Image
  //  Simple forward undistortion implementation (may contain holes)
  //  Input sizes
  //  Allocate output matrices
  //  Apply Radial and Tangential Coeffs
  hipMemcpy(*gpu_rightImage, rightImage, 2073600ULL, hipMemcpyHostToDevice);
  hipMemcpy(*gpu_leftImage, leftImage, 2073600ULL, hipMemcpyHostToDevice);
  generateScene3DGpuImpl_kernel1<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_rightImage, *gpu_leftImage, *gpu_I2U8, *gpu_I1U8);
  //  Rectify Images
  //  Performing inverse mapping and bilinear interpolation
  generateScene3DGpuImpl_kernel2<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_I2U8, *gpu_I1U8, *gpu_rightRect, *gpu_leftRect, 0.990029752F,
      -0.00356712751F, -1.07433525E-5F, -0.00229903217F, 0.999997377F,
      -1.36175793E-16F, 27.5765533F, 3.26931334F, 1.00978231F, 0.998733819F,
      0.000572044635F, -5.7333632E-6F, 0.0028441071F, 1.00773752F,
      6.31935836E-6F, 130.598694F, 0.587008119F, 1.00176048F);
  //  Preprocessing Image
  stencilKernel<<<8160U, 256U>>>(*gpu_leftRect, 0U, *gpu_leftFiltered);
  stencilKernel<<<8160U, 256U>>>(*gpu_rightRect, 0U, *gpu_rightFiltered);
  //  Compute disparity
  generateScene3DGpuImpl_kernel3<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_rightFiltered, *gpu_leftFiltered, *gpu_I2U8, *gpu_I1U8);
  generateScene3DGpuImpl_kernel4<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_I2U8, *gpu_I1U8, *gpu_I4, *gpu_I3);
  computeDisparity(1, 15, 200, 1080, 1920, 5, 0, 40, 0, &(*gpu_I3)[0],
                   &(*gpu_I4)[0], &(*gpu_leftFiltered)[0],
                   &(*gpu_gpuWorkspace)[0]);
  generateScene3DGpuImpl_kernel5<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_leftFiltered, *gpu_disparityMap);
  //  Post processing
  globalConvergenceFlag = true;
  generateScene3DGpuImpl_kernel6<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_disparityMap);
  while (globalConvergenceFlag) {
    globalConvergenceFlag = false;
    hipMemcpy(gpu_globalConvergenceFlag, &globalConvergenceFlag, 1ULL,
               hipMemcpyHostToDevice);
    generateScene3DGpuImpl_kernel11<<<dim3(4041U, 1U, 1U),
                                      dim3(512U, 1U, 1U)>>>(
        gpu_globalConvergenceFlag, *gpu_disparityMap);
    hipMemcpy(&globalConvergenceFlag, gpu_globalConvergenceFlag, 1ULL,
               hipMemcpyDeviceToHost);
  }
  //  Reprojection
  //  Generate Meshgrids
  generateScene3DGpuImpl_kernel7<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_yCoord, *gpu_xCoord);
  //  Stack points
  generateScene3DGpuImpl_kernel8<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_disparityMap, *gpu_yCoord, *gpu_xCoord, *gpu_pts3DHomogeneous);
  //  Reprojection
  generateScene3DGpuImpl_kernel9<<<dim3(16200U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *fv_gpu_clone, *gpu_pts3DHomogeneous, *gpu_pts3DHomogeneousReproject);
  generateScene3DGpuImpl_kernel10<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_disparityMap, *gpu_pts3DHomogeneousReproject, *gpu_pts3DOut);
  //  Points of Cloud
  hipMemcpy(leftRect, *gpu_leftRect, 2073600ULL, hipMemcpyDeviceToHost);
  hipMemcpy(rightRect, *gpu_rightRect, 2073600ULL, hipMemcpyDeviceToHost);
  hipMemcpy(disparityMap, *gpu_disparityMap, 8294400ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(pts3DOut, *gpu_pts3DOut, 24883200ULL, hipMemcpyDeviceToHost);
  mwCudaFree(&(*gpu_rightImage)[0]);
  mwCudaFree(&(*gpu_leftImage)[0]);
  mwCudaFree(&(*gpu_I2U8)[0]);
  mwCudaFree(&(*gpu_I1U8)[0]);
  mwCudaFree(&(*gpu_rightRect)[0]);
  mwCudaFree(&(*gpu_leftRect)[0]);
  mwCudaFree(&(*gpu_leftFiltered)[0]);
  mwCudaFree(&(*gpu_rightFiltered)[0]);
  mwCudaFree(&(*gpu_I4)[0]);
  mwCudaFree(&(*gpu_I3)[0]);
  mwCudaFree(&(*gpu_gpuWorkspace)[0]);
  mwCudaFree(&(*gpu_disparityMap)[0]);
  mwCudaFree(gpu_globalConvergenceFlag);
  mwCudaFree(&(*gpu_yCoord)[0]);
  mwCudaFree(&(*gpu_xCoord)[0]);
  mwCudaFree(&(*gpu_pts3DHomogeneous)[0]);
  mwCudaFree(&(*gpu_pts3DHomogeneousReproject)[0]);
  mwCudaFree(&(*gpu_pts3DOut)[0]);
}

// End of code generation (generateScene3DGpuImpl.cu)
