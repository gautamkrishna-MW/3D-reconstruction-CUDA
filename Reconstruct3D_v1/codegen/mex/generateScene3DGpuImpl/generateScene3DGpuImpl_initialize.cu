//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
//
// generateScene3DGpuImpl_initialize.cu
//
// Code generation for function 'generateScene3DGpuImpl_initialize'
//

// Include files
#include "generateScene3DGpuImpl_initialize.h"
#include "_coder_generateScene3DGpuImpl_mex.h"
#include "generateScene3DGpuImpl_data.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWMemoryManager.hpp"

// Function Declarations
static void generateScene3DGpuImpl_once();

// Function Definitions
static void generateScene3DGpuImpl_once()
{
  mwMemoryManagerInit(256U, 0U, 8U, 2048U);
  mwCudaMalloc(&fv_gpu_clone, static_cast<uint64_T>(sizeof(real32_T[16])));
}

void generateScene3DGpuImpl_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "video_and_image_blockset", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "image_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    generateScene3DGpuImpl_once();
  }
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (generateScene3DGpuImpl_initialize.cu)
